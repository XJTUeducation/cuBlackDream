/**
 * @file   : logreg.cu
 * @brief  : Logistic Regression with CUDA C++14, CUBLAS, CUDA Unified Memory Management
 * @details :  class CSVRow
 * 
 * @author : Ernest Yeung <ernestyalumni@gmail.com>
 * @date   : 20171021  
 * @ref    : Coursera, Andrew Ng, Intro. to Machine Learning, ex1, Exercises 1 of Week 2
 * 
 * https://www.paypal.com/cgi-bin/webscr?cmd=_donations&business=ernestsaveschristmas%2bpaypal%40gmail%2ecom&lc=US&item_name=ernestyalumni&currency_code=USD&bn=PP%2dDonationsBF%3abtn_donateCC_LG%2egif%3aNonHosted 
 * 
 * which won't go through a 3rd. party such as indiegogo, kickstarter, patreon.  
 * Otherwise, I receive emails and messages on how all my (free) material on 
 * physics, math, and engineering have helped students with their studies, 
 * and I know what it's like to not have money as a student, but love physics 
 * (or math, sciences, etc.), so I am committed to keeping all my material 
 * open-source and free, whether or not 
 * sufficiently crowdfunded, under the open-source MIT license: 
 * 	feel free to copy, edit, paste, make your own versions, share, use as you wish.  
 *  Just don't be an asshole and not give credit where credit is due.  
 * Peace out, never give up! -EY
 * 
 * */
/* 
 * COMPILATION TIP
 * nvcc -arch='sm_52' -std=c++14 -lcublas ../src/Feedfwd/Axon.o ../src/Feedfwd/activationf.o ../src/Feedfwd/Feedfwd.o ../src/FileIO/FileIO.cpp logreg.cu -o logreg.exe
 * 
 * */
 
#include <string> 	// std::string

#include "../src/FileIO/FileIO.h"			// csv2fvec, h_flatten_colmaj
#include "../src/Feedfwd/Feedfwd.h"

#include "../src/Axon/activationf.h"

#include <stdio.h> // printf

 
int main(int argc, char* argv[]) { 
	/** ==================== timing CUDA operations ==================== 
	 * @ref https://stackoverflow.com/questions/7876624/timing-cuda-operations
	 * */
	float timeinterval; 
	hipEvent_t starttiming, stoptiming;
	hipEventCreate(&starttiming);
	hipEventCreate(&stoptiming);


	/* =============== ex2data1.txt =============== */ 
	std::string filename_ex2data1 = "../data/ex2data1.txt";
	auto fvec_ex2data1 = csv2fvec( filename_ex2data1) ;
	std::vector<std::vector<float>> X_ex1data2; // choose this type, than std::vector<float> so to generalize to multi-dim. case
	std::vector<std::vector<float>> y_ex1data2;
	for (auto row : fvec_ex2data1) { 
		std::vector<float> X_i = { row[0], row[1] }; // first 2 "columns" is input data X \in \mathbb{R}^m, d=2 features
		std::vector<float> y_i = { row[2] }; // second "column" is output data y \in \mathbb{R}^m, K=1 dim.
		X_ex1data2.push_back(X_i);	
		y_ex1data2.push_back(y_i);	
	}

	int d = X_ex1data2[0].size(); // number of features
	int K = y_ex1data2[0].size(); // dim. of output
	int m = X_ex1data2.size(); 	// m = number of training examples
	
	std::cout << std::endl << " For ex2data1.txt : " << std::endl; 
	std::cout << " d = " << d << ", K = " << K << ", m = " << m << std::endl;

	auto X_ex2data1_colmaj = h_flatten_colmaj(X_ex1data2); 
	auto y_ex2data1_colmaj = h_flatten_colmaj(y_ex1data2); 

	/* sanity check 
	for (auto ele : X_ex2data1_colmaj) { 
	std::cout << ele << " " ; }
	*/
	
	std::vector<int> FFsizeDims = { d,K }; 
	// pick sigmoid function
	std::vector<int> FFactfs = { 1 };

	LogisticReg logreg( FFsizeDims, FFactfs );

	// Initialize fitting parameters with 0
	std::vector<float> h_Theta(d*K, 0.f);
	std::vector<float> h_b(K,0.f );
	std::vector<std::vector<float>> h_Thetab;
	h_Thetab.push_back(h_Theta);
	h_Thetab.push_back(h_b);

	logreg.load_from_hThetaBs( h_Thetab);
	
	// sanity check
	// it WORKS
//	for (auto ele : y_ex2data1_colmaj) { std::cout << ele << " " ; } std::cout << std::endl;

	logreg.load_y_from_hvec(y_ex2data1_colmaj);

	// sanity check
	// it WORKS
/*	auto ycheck = logreg.gety();
	std::vector<float> hycheck(m*K,0.f);
	hipMemcpy(hycheck.data(), ycheck.get(), m*K *sizeof(float),hipMemcpyDeviceToHost);
	for (auto ele : hycheck) { 
		std::cout << ele << " " ; } std::cout << std::endl;
*/

	logreg.load_X_from_hvec(X_ex2data1_colmaj, m);
	// sanity check
/*	auto alm1check = logreg.getalm1(1);
	std::vector<float> halm1check(m*d,0.f);
	hipMemcpy(halm1check.data(), alm1check.get(), m*d *sizeof(float),hipMemcpyDeviceToHost);
	for (auto ele : X_ex2data1_colmaj) { std::cout << ele << " " ; } std::cout << std::endl;
	for (auto ele : halm1check) { 
		std::cout << ele << " " ; } std::cout << std::endl;
*/
	// sanity check
/*	auto alcheck = logreg.getal(1);
	sigmoid_kernel<<<(m*K+128-1)/128,128>>>(m*K, alcheck.get()) ;
	std::vector<float> halcheck(m*K,0.f);
	hipMemcpy(halcheck.data(), alcheck.get(), m*K *sizeof(float),hipMemcpyDeviceToHost);
	for (auto ele : halcheck) { 
		std::cout << ele << " " ; } std::cout << std::endl;
*/

	logreg.feedfwd(256);
	// sanity check
	// it WORKS
/*	auto alcheck = logreg.getal(1);
	std::vector<float> halcheck(m*K,0.f);
	hipMemcpy(halcheck.data(), alcheck.get(), m*K *sizeof(float),hipMemcpyDeviceToHost);
	for (auto ele : halcheck) { 
		std::cout << ele << " " ; } std::cout << std::endl;
*/

	// sanity check
	float result_logregcost = 0.f; 
	result_logregcost = logreg.compute_costJ_xent(128);
	std::cout << " costJ for cross-entropy function, at initial theta (zeros) : " << result_logregcost << std::endl;
	std::cout << " Expected cost (approx): 0.693" << std::endl; 

	logreg.grad_desc_step(0.01f, 256);

	/* sanity check of 1 gradient descent  
	 * this (block of code) WORKS
	 * */
/*	auto Theta1 = std::move( logreg.getTheta(1) );
	auto b1 = std::move( logreg.getb(1) );
	std::vector<float> hTheta1(d*K,-1.f);
	std::vector<float> hb1(K,-1.f);
	hipMemcpy(hTheta1.data(), Theta1.get(), d*K*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(hb1.data(), b1.get(), K*sizeof(float), hipMemcpyDeviceToHost);
	std::cout << " hTheta1 : " << hTheta1[0] << " " << hTheta1[1] << std::endl;
	std::cout << " hb1 : " << hb1[0] << " " << std::endl;
*/

	/* ========== Compute and display cost and gradient with non-zero theta ========== */
	std::vector<float> h_testTheta { 0.2f, 0.2f }; 
	std::vector<float> h_testb { -24.f };
	std::vector<std::vector<float>> h_testThetab;
	h_testThetab.push_back(h_testTheta);
	h_testThetab.push_back(h_testb);

	logreg.load_from_hThetaBs( h_testThetab);
	logreg.feedfwd(256);
	float result_testlogregcost = logreg.compute_costJ_xent(256);

	std::cout << std::endl << " Cost at test theta: " << result_testlogregcost << std::endl;
	std::cout << " Expected cost (approx): 0.218 " << std::endl; 

	logreg.grad_desc_step(0.01f, 256);

	/* sanity check of 1 gradient descent  
	 * this (block of code) WORKS
	 * */
/*	auto Theta1 = std::move( logreg.getTheta(1) );
	auto b1 = std::move( logreg.getb(1) );
	std::vector<float> hTheta1(d*K,-1.f);
	std::vector<float> hb1(K,-1.f);
	hipMemcpy(hTheta1.data(), Theta1.get(), d*K*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(hb1.data(), b1.get(), K*sizeof(float), hipMemcpyDeviceToHost);
	std::cout << " hTheta1 : " << hTheta1[0] << " " << hTheta1[1] << std::endl;
	std::cout << " hb1 : " << hb1[0] << " " << std::endl;
*/
	// fprintf('Expected gradients (approx):\n 0.043\n 2.566\n 2.647\n');
	logreg.load_from_hThetaBs( h_Thetab);
	logreg.feedfwd(256);


	hipEventRecord(starttiming,0);
	logreg.grad_desc(1500000,0.0010f, 256);
	hipEventRecord(stoptiming,0);
	hipEventSynchronize(stoptiming);
	hipEventElapsedTime(&timeinterval, starttiming,stoptiming);
	printf("Time to grad_desc 1500 iterations : %3.1f ms \n ", timeinterval);

	result_testlogregcost = logreg.compute_costJ_xent(256);


	std::cout << " Cost at theta found by grad desc: " << result_testlogregcost << std::endl; 
	std::cout << " Expected cost (approx): 0.203 " << std::endl; 


	/* sanity check of gradient descent  
	 * this (block of code) WORKS
	 * */
	auto Theta1 = std::move( logreg.getTheta(1) );
	auto b1 = std::move( logreg.getb(1) );
	std::vector<float> hTheta1(d*K,0.f);
	std::vector<float> hb1(K,0.f);
	hipMemcpy(hTheta1.data(), Theta1.get(), d*K*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(hb1.data(), b1.get(), K*sizeof(float), hipMemcpyDeviceToHost);
	std::cout << " hTheta1 : " << hTheta1[0] << " " << hTheta1[1] << std::endl;
	std::cout << " hb1 : " << hb1[0] << " " << std::endl;

	// fprintf('Expected theta (approx):\n');
	// fprintf(' -25.161\n 0.206\n 0.201\n');



}

