#include "hip/hip_runtime.h"
/**
 * @file   : Axon.cu
 * @brief  : Axon content/source file in CUDA C++14, 
 * @author : Ernest Yeung <ernestyalumni@gmail.com>
 * @date   : 20171007  
 * 
 * If you find this code useful, feel free to donate directly and easily at this direct PayPal link: 
 * 
 * https://www.paypal.com/cgi-bin/webscr?cmd=_donations&business=ernestsaveschristmas%2bpaypal%40gmail%2ecom&lc=US&item_name=ernestyalumni&currency_code=USD&bn=PP%2dDonationsBF%3abtn_donateCC_LG%2egif%3aNonHosted 
 * 
 * which won't go through a 3rd. party such as indiegogo, kickstarter, patreon.  
 * Otherwise, I receive emails and messages on how all my (free) material on 
 * physics, math, and engineering have helped students with their studies, 
 * and I know what it's like to not have money as a student, but love physics 
 * (or math, sciences, etc.), so I am committed to keeping all my material 
 * open-source and free, whether or not 
 * sufficiently crowdfunded, under the open-source MIT license: 
 * 	feel free to copy, edit, paste, make your own versions, share, use as you wish.  
 *  Just don't be an asshole and not give credit where credit is due.  
 * Peace out, never give up! -EY
 * 
 * */
/* 
 * COMPILATION TIP
 * nvcc -std=c++14 -lcublas -dc Axon.cu -o Axon.o
 * 
 * */
#include "Axon.h"
#include "activationf.h"

/* =============== CUDA kernel functions =============== */

/** @fn setconstval_kernel
 * 	@brief set a float array of length Lx all to values of const_val 
 * 	@details hipMemset only sets an array to 0 value; we want value of 1
 * */
__global__ void setconstval_kernel(const int Lx, const float const_val, float* A) {
	int kx = threadIdx.x + blockDim.x * blockIdx.x; 
	if (kx >= Lx) { 
		return ; 
	} 

	for (int tid=kx; tid < Lx; tid += gridDim.x * blockDim.x ) {
		A[tid] = const_val ; 	
	}
}


/* ==================== Axon classes ==================== */

/* =============== Axon class; no activation =============== */


// constructor 
Axon::Axon(const int s_lm1,const int s_l) : s_lm1(s_lm1), s_l(s_l)  {
	const int SIZE_THETA = s_l*s_lm1;

	std::unique_ptr<float[], deleterRR_struct> d_Theta(new float[SIZE_THETA]);
	hipMallocManaged((void **) &d_Theta,SIZE_THETA*sizeof(float));
	Theta = std::move(d_Theta);

	std::unique_ptr<float[], deleterRR_struct> d_b(new float[s_l]);
	hipMallocManaged((void **) &d_b,s_l*sizeof(float));
	b = std::move(d_b);
}

// Move Constructor
/**
 *  @fn Axon(const Axon& old_axon)
 *  @brief copy constructor for Axon class
 * 	@ref http://www.geeksforgeeks.org/copy-constructor-in-cpp/
 * https://stackoverflow.com/questions/16030081/copy-constructor-for-a-class-with-unique-ptr
 * https://en.wikipedia.org/wiki/C%2B%2B11#Rvalue_references_and_move_constructors
 * */
Axon::Axon(Axon&& old_axon) : Theta(std::move(old_axon.Theta)), b(std::move(old_axon.b))
{
	s_lm1 = old_axon.s_lm1;
	s_l = old_axon.s_l;
	m = old_axon.m;
	
	l = old_axon.l; // lth layer
	
	alm1 = std::move( old_axon.alm1 );
	al = std::move( old_axon.al );	
}

// operator overload assignment = 
Axon & Axon::operator=(Axon && old_axon) {
	s_lm1 = old_axon.s_lm1;
	s_l = old_axon.s_l;
	m = old_axon.m;
	
	l = old_axon.l; // lth layer

	// shared_ptrs moved
	alm1 = std::move( old_axon.alm1 );
	al = std::move( old_axon.al );	

	// unique_ptrs moved
	Theta = std::move(old_axon.Theta);
	b = std::move( old_axon.b );

	return *this;
}

// member functions
void Axon::load_from_hvec(std::vector<float>& h_Theta,std::vector<float>& h_b) {
	const int SIZE_THETA = s_l*s_lm1;

	hipMemcpy(Theta.get(), h_Theta.data(), SIZE_THETA*sizeof(float),hipMemcpyHostToDevice);	
	hipMemcpy(b.get(), h_b.data(), s_l*sizeof(float),hipMemcpyHostToDevice);	
}	

/**
 * 	@fn load_from_d 
 * 	@brief (Theta,b) on device GPU -> std::vector on host 
 * */
void Axon::load_from_d(std::vector<float>& h_Theta, std::vector<float>& h_b) {
	const int SIZE_THETA = s_l*s_lm1;

	hipMemcpy(h_Theta.data(), Theta.get(), SIZE_THETA*sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(h_b.data(), b.get(), s_l*sizeof(float),hipMemcpyDeviceToHost);

}		

// for loading input data X into layer l-1, alm1
/**
 * 	@fn load_from_hXvec 
 * 	@brief load from host, X input data, as a std::vector<float>
 *  @param const int m - number of examples
 * */
void Axon::load_from_hXvec(std::vector<float>& h_X, const int m) {
	const int SIZE_S_LM1 = m * s_lm1;
	
	if (!alm1.get()) {
		std::shared_ptr<float> d_alm1(new float[SIZE_S_LM1], deleterRR_struct() ); 	// d_alm1; alm1 on device GPU
		hipMallocManaged((void **) &d_alm1,SIZE_S_LM1*sizeof(float));
		alm1 = std::move(d_alm1);
		hipMemcpy(alm1.get(), h_X.data(), SIZE_S_LM1 *sizeof(float),hipMemcpyHostToDevice);
		
	} else {
		hipMemcpy(alm1.get(), h_X.data(), SIZE_S_LM1 *sizeof(float),hipMemcpyHostToDevice);
	}
	this->m = m;
}

	/** We're not transferring ownership, so we don't use std::move
	 * @ref https://stackoverflow.com/questions/41871115/why-would-i-stdmove-an-stdshared-ptr
	 * */
void Axon::load_alm1_from_ptr(std::shared_ptr<float> & ptr_sh_input_layer) 
{
	alm1 = ptr_sh_input_layer;
}

/** We're transferring ownership, so we  use std::move
  * @ref https://stackoverflow.com/questions/41871115/why-would-i-stdmove-an-stdshared-ptr
  * */
void Axon::move2al_from_ptr(std::shared_ptr<float> & ptr_sh_output_layer) 
{
	al = std::move( ptr_sh_output_layer );
}

void Axon::move2alm1_from_ptr(std::shared_ptr<float> & ptr_sh_input_layer) 
{
	alm1 = std::move( ptr_sh_input_layer );
}


// initialize layer l
/**
 * 	@fn init_al 
 * 	@brief initialize layer l
 *  @param const int m - number of examples
 * */
void Axon::init_al(const int m) { 
	const int SIZE_S_L = m * s_l;

	std::shared_ptr<float> d_al(new float[SIZE_S_L], deleterRR_struct() ); 	// d_al; al on device GPU
	hipMallocManaged((void **) &d_al,SIZE_S_L*sizeof(float));
	al = std::move(d_al);
	hipMemset(al.get(), 0.f, SIZE_S_L*sizeof(float));

	this->m = m;
}



// for getting size dimensions
std::vector<int> Axon::getSizeDims() {
	std::vector<int> sizedimsvec = { s_lm1, s_l, m };
	return sizedimsvec;
}


// for getting Theta,b, and lth layer al, zl (after activation function applied)

std::unique_ptr<float[], deleterRR_struct> Axon::getTheta() {
	auto ptr = std::move(Theta);
	return ptr;
}

std::unique_ptr<float[],deleterRR_struct> Axon::getb() {
	auto ptr = std::move(b);
	return ptr;
}

void Axon::move2Theta_from_ptr(std::unique_ptr<float[], deleterRR_struct> & ptr_Theta) 
{
	Theta = std::move( ptr_Theta );
}

void Axon::move2b_from_ptr(std::unique_ptr<float[], deleterRR_struct> & ptr_b) 
{
	b = std::move( ptr_b );
}

/**
 * @fn Axon::getalm1
 * @details we don't use std::move because we don't want to change (move) 
 * 	ownership of the pointer (and the memory it points to) because we're 
 *  dealing with a shared_ptr (you can move it, but then we'd want to use a 
 * 	unique_ptr; we want to share it)
 * */
std::shared_ptr<float> Axon::getalm1() {
	auto ptr = alm1;
	return ptr;
}

/**
 * @fn Axon::getal
 * @details we don't use std::move because we don't want to change (move) 
 * 	ownership of the pointer (and the memory it points to) because we're 
 *  dealing with a shared_ptr (you can move it, but then we'd want to use a 
 * 	unique_ptr; we want to share it)
 * */
std::shared_ptr<float> Axon::getal() {
	auto ptr = al;
	return ptr;
}


/* =============== "connect" the Axon =============== */
/* Once Axon has been setup, by the above, do the following to 
/* "connect through" the Axon */

/**
 *  @fn rightMul
 *  @class Axon_
 * 	@brief right multiplication
 * */
void Axon::rightMul() {
	float a1 = 1.0f;
	float bet = 0.f;
	
	std::unique_ptr<hipblasHandle_t,del_cublasHandle_struct> handle_u(
		new hipblasHandle_t);
	hipblasCreate(handle_u.get());
		
	hipblasSgemm(*handle_u.get(),HIPBLAS_OP_N,HIPBLAS_OP_N,m,s_l,s_lm1,
		&a1, alm1.get(),m, Theta.get(),s_lm1,&bet,al.get(),m);

}

	/* ========== Add bias ========== */
/** 
 * 	@fn Axon::addb
 * 	@param const int N_x = number of (thread) blocks on grid in x-direction
 *  @param const int M_x = number of threads in a (single, thread) block in x-direction
 * 	@details N_x, M_x determined before by feedfwd class
 * */
void Axon::addb(const int M_x,const int N_x ) {

	/* ===== grid, thread block size dimensions ===== */
	const int SIZE_A_L = m * s_l; // m * s_l = (number of examples)*(size dim. or no. of nodes of lth layer)
	
	// M_x = number of threads in a (single) block in x-direction
	int Nx = 0;
	if (N_x == 0) { 
		const int Nx_calc = (SIZE_A_L + M_x -1)/M_x;
		Nx = max( Nx_calc, N_x);
	} else {
		Nx = N_x;
	}

	// create 1s array, array of 1s
	std::unique_ptr<float[], deleterRR_struct> ones(new float[SIZE_A_L], deleterRR_struct());
	hipMallocManaged((void **) &ones, SIZE_A_L*sizeof(float));

	setconstval_kernel<<<Nx,M_x>>>(SIZE_A_L,1.0f, ones.get() );

	// create "broadcasted" array for bias b
	std::unique_ptr<float[], deleterRR_struct> broadcast_b(new float[SIZE_A_L], deleterRR_struct());
	hipMallocManaged((void **) &broadcast_b, SIZE_A_L*sizeof(float));

	std::unique_ptr<hipblasHandle_t,del_cublasHandle_struct> handle_u(
		new hipblasHandle_t);
	hipblasCreate(handle_u.get());	

	// C = A x diag(X)
	hipblasSdgmm( *handle_u.get(), HIPBLAS_SIDE_RIGHT, 
		m, s_l, ones.get(), m, b.get(), 1, broadcast_b.get(), m);

	// C = \alpha op(A) + \beta op(B)
	float a1 = 1.0f; 
	float bet = 1.0f;
	hipblasSgeam(*handle_u.get(), HIPBLAS_OP_N, HIPBLAS_OP_N, 
		m, s_l, 
		&a1, broadcast_b.get(), m, &bet, al.get(), m, 
		al.get(), m);


}
	


// destructor
Axon::~Axon() {}


/* =============== Axon class; with activation =============== */
// Constructor
Axon_act::Axon_act(const int s_lm1,const int s_l, const int idx_actf) : 
		Axon(s_lm1, s_l), idx_actf(idx_actf) { }

// Move Constructor
/**
 *  @fn Axon_act(const Axon& old_axon)
 *  @brief copy constructor for Axon class
 * 	@ref http://www.geeksforgeeks.org/copy-constructor-in-cpp/
 * https://stackoverflow.com/questions/16030081/copy-constructor-for-a-class-with-unique-ptr
 * https://en.wikipedia.org/wiki/C%2B%2B11#Rvalue_references_and_move_constructors
 * https://msdn.microsoft.com/en-us/library/s16xw1a8.aspx
 * */
Axon_act::Axon_act(Axon_act&& old_axon) 
	: 	Axon(std::move(old_axon)), // error: function "Axon::Axon(const Axon &)" (declared implicitly) cannot be referenced -- it is a deleted function

	 zl(std::move(old_axon.zl)),
	 Dpsil(std::move(old_axon.Dpsil))
{
	idx_actf = old_axon.idx_actf;
}


// operator overload assignment = 
Axon_act & Axon_act::operator=(Axon_act && old_axon) 
{

	idx_actf = old_axon.idx_actf;

	zl = std::move( old_axon.zl );
	Dpsil = std::move( old_axon.Dpsil);

	return *this;
}

// initialize layer l
/**
 * 	@fn init_zlal 
 * 	@brief initialize layer l
 *  @param const int m - number of examples
 * */
void Axon_act::init_zlal(const int m) { 
	const int SIZE_S_L = m * s_l;

	std::shared_ptr<float> d_al(new float[SIZE_S_L], deleterRR_struct() ); 	// d_al; al on device GPU
	hipMallocManaged((void **) &d_al,SIZE_S_L*sizeof(float));
	al = std::move(d_al);
	hipMemset(al.get(), 0.f, SIZE_S_L*sizeof(float));

	std::unique_ptr<float[], deleterRR_struct> d_zl(new float[SIZE_S_L], deleterRR_struct());
	hipMallocManaged((void **) &d_zl,SIZE_S_L*sizeof(float));
	zl = std::move(d_zl);

	this->m = m;
}

// for getting Theta,b, and lth layer al, zl (after activation function applied)

std::unique_ptr<float[],deleterRR_struct> Axon_act::getzl() {
	auto ptr = std::move(zl);
	return ptr;
}

std::unique_ptr<float[],deleterRR_struct> Axon_act::getDpsil() {
	auto ptr = std::move(Dpsil);
	return ptr;
}


/* =============== "connect" the Axon =============== */
/* Once Axon has been setup, by the above, do the following to 
/* "connect through" the Axon */
/**
 *  @fn rightMul
 *  @class Axon_act
 * 	@brief right multiplication
 * */
void Axon_act::rightMul() {
	float a1 = 1.0f;
	float bet = 0.f;
	
	std::unique_ptr<hipblasHandle_t,del_cublasHandle_struct> handle_u(
		new hipblasHandle_t);
	hipblasCreate(handle_u.get());
	
	hipblasSgemm(*handle_u.get(),HIPBLAS_OP_N,HIPBLAS_OP_N,m,s_l,s_lm1,
		&a1,alm1.get(),m,Theta.get(),s_lm1,&bet,zl.get(),m);


}



	/* ========== Add bias ========== */
/** 
 * 	@fn Axon_act::addb
 * 	@param const int N_x = number of (thread) blocks on grid in x-direction
 *  @param const int M_x = number of threads in a (single, thread) block in x-direction
 * 	@details N_x, M_x determined before by feedfwd class
 * */
void Axon_act::addb(const int M_x, const int N_x) {

	/* ===== grid, thread block size dimensions ===== */
	const int SIZE_Z_L = m * s_l; // m * s_l = (number of examples)*(size dim. or no. of nodes of lth layer)
	
	// M_x = number of threads in a (single) block in x-direction
	int Nx = 0;
	if (N_x == 0) { 
		const int Nx_calc = (SIZE_Z_L + M_x -1)/M_x;
		Nx = max( Nx_calc, N_x);
	} else {
		Nx = N_x;
	}

	// create 1s array, array of 1s
	std::unique_ptr<float[], deleterRR_struct> ones(new float[SIZE_Z_L], deleterRR_struct());
	hipMallocManaged((void **) &ones, SIZE_Z_L*sizeof(float));

	setconstval_kernel<<<Nx,M_x>>>(SIZE_Z_L,1.0f, ones.get() );

	// create "broadcasted" array for bias b
	std::unique_ptr<float[], deleterRR_struct> broadcast_b(new float[SIZE_Z_L], deleterRR_struct());
	hipMallocManaged((void **) &broadcast_b, SIZE_Z_L*sizeof(float));

	std::unique_ptr<hipblasHandle_t,del_cublasHandle_struct> handle_u(
		new hipblasHandle_t);
	hipblasCreate(handle_u.get());	

	// C = A x diag(X)
	hipblasSdgmm( *handle_u.get(), HIPBLAS_SIDE_RIGHT, 
		m, s_l, ones.get(), m, b.get(), 1, broadcast_b.get(), m);

	// C = \alpha op(A) + \beta op(B)
	float a1 = 1.0f; 
	float bet = 1.0f;
	hipblasSgeam(*handle_u.get(), HIPBLAS_OP_N, HIPBLAS_OP_N, 
		m, s_l, 
		&a1, broadcast_b.get(), m, &bet, zl.get(), m, 
		zl.get(), m);

}

/* ========== activate with activation function ========== */
void Axon_act::actf( const int M_x, const int N_x) {

	/* ===== grid, thread block size dimensions ===== */
	const int SIZE_Z_L = m * s_l; // m * s_l = (number of examples)*(size dim. or no. of nodes of lth layer)

	hipMemcpy(al.get(), zl.get(), sizeof(float) * SIZE_Z_L, hipMemcpyDeviceToDevice) ; 
	
	// M_x = number of threads in a (single) block in x-direction
	int Nx = 0;
	if (N_x == 0) { 
		const int Nx_calc = (SIZE_Z_L + M_x -1)/M_x;
		Nx = max( Nx_calc, N_x);
	} else {
		Nx = N_x;
	}

	/** using array of function ptr doesn't work because it has to be located to device code and, refer here: 
	 * @ref http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#function-pointers
	 * https://devtalk.nvidia.com/default/topic/457094/cuda-programming-and-performance/how-can-i-use-__device__-function-pointer-in-cuda-/3
	 * https://stackoverflow.com/questions/15644261/cuda-function-pointers/15646771#15646771
	general_activation_function_kernel<<<Nx,M_x>>>( SIZEDIM_Z_L, ptr_zl.get(), idx_actf );
	*/
	if (idx_actf==0) {
		identity_kernel<<<Nx,M_x>>>(SIZE_Z_L, al.get()); 
	} else if (idx_actf==1) {
		sigmoid_kernel<<<Nx,M_x>>>(SIZE_Z_L, al.get() );
	} else if (idx_actf==2) {
		tanh_kernel<<<Nx,M_x>>>(SIZE_Z_L, al.get() );
	} else if (idx_actf==3) {
		tanh_kernel<<<Nx,M_x>>>(SIZE_Z_L, al.get() );
	} else if (idx_actf==4) {
		arctan_kernel<<<Nx,M_x>>>(SIZE_Z_L, al.get() );
	} else if (idx_actf==5) {
		ReLU_kernel<<<Nx,M_x>>>(SIZE_Z_L, al.get() );
	}	


} 


/* ========== partial derivatives with respect to z^l of psi^l(z^l) ========== */
void Axon_act::do_Dpsi( const int M_x, const int N_x) {
	// initialize (i.e. instantiate, construct) 
	const int SIZE_Z_L = m * s_l;

	std::unique_ptr<float[], deleterRR_struct> d_Dpsi(new float[SIZE_Z_L], deleterRR_struct());
	hipMallocManaged((void **) &d_Dpsi,SIZE_Z_L*sizeof(float));


	/* ===== grid, thread block size dimensions ===== */
	hipMemcpy(d_Dpsi.get(), zl.get(), sizeof(float) * SIZE_Z_L, hipMemcpyDeviceToDevice) ; 
	
	// M_x = number of threads in a (single) block in x-direction
	int Nx = 0;
	if (N_x == 0) { 
		const int Nx_calc = (SIZE_Z_L + M_x -1)/M_x;
		Nx = max( Nx_calc, N_x);
	} else {
		Nx = N_x;
	}

	/** using array of function ptr doesn't work because it has to be located to device code and, refer here: 
	 * @ref http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#function-pointers
	 * https://devtalk.nvidia.com/default/topic/457094/cuda-programming-and-performance/how-can-i-use-__device__-function-pointer-in-cuda-/3
	 * https://stackoverflow.com/questions/15644261/cuda-function-pointers/15646771#15646771
	general_activation_function_kernel<<<Nx,M_x>>>( SIZEDIM_Z_L, ptr_zl.get(), idx_actf );
	*/
	if (idx_actf==0) {
		D_identity_kernel<<<Nx,M_x>>>(SIZE_Z_L, zl.get(), d_Dpsi.get() );
	} else if (idx_actf==1) {
		D_sigmoid_kernel<<<Nx,M_x>>>(SIZE_Z_L, zl.get(), d_Dpsi.get() );
	} else if (idx_actf==2) {
		D_tanh_kernel<<<Nx,M_x>>>(SIZE_Z_L, zl.get(), d_Dpsi.get() );
	} else if (idx_actf==3) {
		D_tanh_kernel<<<Nx,M_x>>>(SIZE_Z_L, zl.get(), d_Dpsi.get() );
	} else if (idx_actf==4) {
		D_arctan_kernel<<<Nx,M_x>>>(SIZE_Z_L, zl.get(), d_Dpsi.get() );
	} else if (idx_actf==5) {
		D_ReLU_kernel<<<Nx,M_x>>>(SIZE_Z_L, zl.get(), d_Dpsi.get() );
	}	

	// Remember to move ptr_zl back to zl
	Dpsil = std::move(d_Dpsi);

}


