#include "hip/hip_runtime.h"
/**
 * @file   : DNN.cu
 * @brief  : DNN content/source file in CUDA C++14, 
 * @author : Ernest Yeung <ernestyalumni@gmail.com>
 * @date   : 20171031  
 * @ref    :  
 * 
 * If you find this code useful, feel free to donate directly and easily at this direct PayPal link: 
 * 
 * https://www.paypal.com/cgi-bin/webscr?cmd=_donations&business=ernestsaveschristmas%2bpaypal%40gmail%2ecom&lc=US&item_name=ernestyalumni&currency_code=USD&bn=PP%2dDonationsBF%3abtn_donateCC_LG%2egif%3aNonHosted 
 * 
 * which won't go through a 3rd. party such as indiegogo, kickstarter, patreon.  
 * Otherwise, I receive emails and messages on how all my (free) material on 
 * physics, math, and engineering have helped students with their studies, 
 * and I know what it's like to not have money as a student, but love physics 
 * (or math, sciences, etc.), so I am committed to keeping all my material 
 * open-source and free, whether or not 
 * sufficiently crowdfunded, under the open-source MIT license: 
 * 	feel free to copy, edit, paste, make your own versions, share, use as you wish.  
 *  Just don't be an asshole and not give credit where credit is due.  
 * Peace out, never give up! -EY
 * 
 * */
/* 
 * COMPILATION TIP
 * nvcc -std=c++14 -lcublas -dc Feedfwd.cu -o Feedfwd.o
 * nvcc -std=c++14 -arch='sm_52' -dc ../Axon/Axon.cu ../Axon/activationf.cu Feedfwd.cu
 * */
#include "DNN.h"


/* =============== CUDA functions =============== */

/* =============== CUDA kernel functions =============== */

__global__ void costJ_xent_kernel(const int Lx, const float* y, const float* yhat, float* s) {
	int kx = threadIdx.x + blockDim.x*blockIdx.x;
	if (kx >= Lx) { return; } // more than enough threads were launched to calculate the Lx elements
	
	/* this for loop ensure that, if in case SIZE > gridDimx.x*blockDim.x (or even if  
	 * 	SIZE >> gridDimx.x*blockDim.x so we have more values to compute than threads on a GPU!)  
	 * that everything gets computed) */
	for (int tid=kx; tid < Lx; tid += gridDim.x*blockDim.x ) { 
		float y_val = y[tid];
		float yhat_val = yhat[tid]; 
		float s_val = - y_val * logf( yhat_val) - (1.f - y_val) * logf( 1.f - yhat_val);
		s[tid] = s_val;
	}
}

/**
 * @fn Deltaxent_kernel, __global__ void Deltaxent_kernel
 * @brief compute Delta for the so-called cross-entropy loss function
 * @details Compute
 * ( \widehat{y}^k_{(i)} - y_{(i)}^k )/ (\widehat{y}^k_{(i)} (1 - \widehat{y}_{(i)}^k ) ) 
*/
__global__ void Deltaxent_kernel(const int Lx, const float* y, const float* yhat, float* Delta) {
	int kx = threadIdx.x + blockDim.x*blockIdx.x;
	if (kx >= Lx) { return; } // more than enough threads were launched to calculate the Lx elements
	
	/* this for loop ensure that, if in case SIZE > gridDimx.x*blockDim.x (or even if  
	 * 	SIZE >> gridDimx.x*blockDim.x so we have more values to compute than threads on a GPU!)  
	 * that everything gets computed) */
	for (int tid=kx; tid < Lx; tid += gridDim.x*blockDim.x ) { 
		float y_val = y[tid];
		float yhat_val = yhat[tid]; 
		float Delta_ik = (yhat_val - y_val)/(yhat_val*(1.0f - yhat_val)); 
		Delta[tid] = Delta_ik;
	}
}

/**
 * 	@fn HadamardMultiply
 * 	@brief element-wise multiply  
 * */
__global__ void HadamardMultiply_kernel(const int SIZE, const float* A, float* B) {
	int kx = threadIdx.x + blockDim.x*blockIdx.x;
	if (kx >= SIZE) { return; } // more than enough threads were launched to calculate the Lx elements
	
	/* this for loop ensure that, if in case SIZE > gridDimx.x*blockDim.x (or even if  
	 * 	SIZE >> gridDimx.x*blockDim.x so we have more values to compute than threads on a GPU!)  
	 * that everything gets computed) */
	for (int tid=kx; tid < SIZE; tid += gridDim.x*blockDim.x ) { 
		float A_val = A[tid];
		float B_val = B[tid];
		float C_val = A_val * B_val; 
		B[tid] = C_val;
	}
}


/* ==================== Deep Neural Network (DNN) class ==================== */

/**	@class DNN
 * 	@brief Deep Neural Network (DNN; i.e. Artificial Neural Network (ANN), 
 * 		i.e. so-called "Fully Connected layers")
 * */


// Constructors
DNN::DNN(std::vector<int> & sizeDimsvec, std::vector<int> & actfs_intvec, 
	const int idx_device) : 
	sizeDimsvec(sizeDimsvec), actfs_intvec(actfs_intvec)
{
	const int Lp1 = sizeDimsvec.size(); // L=total number of Axons and so L+1 is total number of layers
	for (int l=1; l<Lp1; l++)  // l= lth Axon
	{
		int s_lm1 = sizeDimsvec[l-1];
		int s_l = sizeDimsvec[l];
		int idx_actf = actfs_intvec[l-1];
	//	Axons.push_back(  Axon_act(s_lm1,s_l,idx_actf, idx_device)  );
	// Don't copy; move.  cf. https://stackoverflow.com/questions/11572669/move-with-vectorpush-back 
		Axons.push_back( std::move( Axon_act(s_lm1,s_l,idx_actf, idx_device) ) );
	}	
	
	// get maximum grid dimension on the device, numbered idx_device (usually 0th device GPU)
	MAX_SIZE_1DARR = get_max_device_array_size1d(idx_device);
	
}
		
// member functions

// for loading (Theta,B) values from host
void DNN::load_from_hThetaBs(std::vector<std::vector<float>> & hThetaBs) {
	const int Lp1 = sizeDimsvec.size(); // L = total number of Axons and so L+1 is total number of layers
	assert (hThetaBs.size()/2 == (Lp1-1) );	// sanity check with input is correct
	for (int l=1; l<Lp1; l++)  // l= lth Axon, l=1,2...L
	{	
		int idx_axon = l-1; // idx_axon=0,1,...L-1
		int idx_Theta = idx_axon*2;
		int idx_b 	  = idx_axon*2 + 1;
		
		Axons[idx_axon].load_from_hvec( hThetaBs[idx_Theta], hThetaBs[idx_b] );
	}
}

// for loading output data y 
/**
 * 	@fn load_y_from_hvec 
 * 	@brief load from host, y output data, as a std::vector<float>, column-major ordered
 * */		
void DNN::load_y_from_hvec(std::vector<float>& h_yvec) {
	const int SIZE_Y= h_yvec.size(); 
	
	std::unique_ptr<float[], deleterRR_struct> d_y(new float[SIZE_Y], deleterRR_struct());
	hipMallocManaged((void **) &d_y, SIZE_Y*sizeof(float));
	y = std::move(d_y);

	hipMemcpy(y.get(), h_yvec.data(), SIZE_Y*sizeof(float),hipMemcpyHostToDevice);	
	
}

// for loading input data X into layer 0, a0, input layer
/**
 * 	@fn load_X_from_hvec
 * 	@brief load from host, X input data, as a std::vector<float>
 * 			Since we're then given m, number of examples (in dataset), 
 * 			load_X_from_hvec will then and go ahead and point (using std::shared_ptr)
 * 			output layer l-1 of Axon l-1 to input layer l-1 of Axon l
 *  @param const int m - number of examples
 * */		
void DNN::load_X_from_hvec(std::vector<float>& h_Xvec, const int m) 
{
	const int SIZE_X = h_Xvec.size();
	const int d = sizeDimsvec[0];
	assert( SIZE_X == m*d); // check the total size dimensions is correct for "input layer" 
	
	// first Axon
	Axons[0].load_from_hXvec( h_Xvec, m);
	Axons[0].init_zlal(m);

	const int Lp1 = sizeDimsvec.size(); // L=total number of Axons and so L+1 is total number of layers
	
	for (int l=2;l<Lp1; l++) {
		int idx_axon = l-1; // l=2,3...L, idx_axon=1,2,...L-1

		// if we didn't assign the shared_ptr and so "share" it, 
		// must do move for 1 command immediately below, because otherwise, error: initial value of reference 
		// to non-const must be an lvalue
		auto tempshptr = Axons[idx_axon-1].getal(); // temporary shared pointer, share, NOT move, ownership to it temporarily
		Axons[idx_axon].load_alm1_from_ptr( tempshptr );
		Axons[idx_axon].init_zlal(m);
//		Axons[idx_axon-1].move2al_from_ptr( tempshptr); // move ownership back to al from temporary shared ptr
		tempshptr.reset();
	}
	this->m=m; // store the number of training examples
}


/* =============== "getting" functions =============== */

// for getting Theta,b, and lth layer of lth Axon al, zl (after activation function applied)

std::unique_ptr<float[], deleterRR_struct> DNN::getTheta(const int l) {
	const int Lp1 = sizeDimsvec.size(); // L = total number of Axons and so L+1 is total number of layers
	assert (l < Lp1);	// sanity check that l=1,2,...L
	int idx_axon = l-1; // ind_axon=0,1,...L-1, 0-based counting
	auto ptr = std::move( Axons[idx_axon].getTheta() );

	return ptr;
}

std::unique_ptr<float[],deleterRR_struct> DNN::getb(const int l) {
	const int Lp1 = sizeDimsvec.size(); // L = total number of Axons and so L+1 is total number of layers
	assert (l < Lp1);	// sanity check that l=1,2,...L
	int idx_axon = l-1; // ind_axon=0,1,...L-1, 0-based counting
	auto ptr = std::move( Axons[idx_axon].getb() );

	return ptr;
}

std::shared_ptr<float> DNN::getalm1(const int l) {
	const int Lp1 = sizeDimsvec.size(); // L = total number of Axons and so L+1 is total number of layers
	assert (l < Lp1);	// sanity check that l=1,2,...L
	int idx_axon = l-1; // ind_axon=0,1,...L-1, 0-based counting
	auto ptr = Axons[idx_axon].getalm1();

	return ptr;
}


std::shared_ptr<float> DNN::getal(const int l) {
	const int Lp1 = sizeDimsvec.size(); // L = total number of Axons and so L+1 is total number of layers
	assert (l < Lp1);	// sanity check that l=1,2,...L
	int idx_axon = l-1; // ind_axon=0,1,...L-1, 0-based counting
	auto ptr = Axons[idx_axon].getal();

	return ptr;
}

std::unique_ptr<float[],deleterRR_struct> DNN::gety() {
	auto ptr = std::move(y);
	return ptr;
}

/* ========== Feedforward ========== */
/**
 *  @fn feedfwd
 * 	@brief Feedforward
 * 	@param Mx, int Mx=128, default to 128 threads in a single thread block
 * 		when adding the bias to the output layer of an axon, choose the number of threads in a single 
 * */

void DNN::feedfwd(int M_x) {
	const int Lp1 = sizeDimsvec.size(); // L = total number of Axons and so L+1 is total number of layers

	for (int l=1; l < Lp1;l++) {
		int idx_axon = l-1; // l=1,2,...L axons, idx_axon = 0,1,...L-1 (0-based counting for C/C++/Python
		Axons[idx_axon].rightMul();	// a^{l-1} \Theta = (a^{l-1}_i)^{j_{l-1}} \Theta_{j_{l-1}}^{j_l} =: z^l 
		Axons[idx_axon].addb(M_x);	// z^l +b = (z^l_i)^{j_l} + (b^{(l)})^{j_l} =: z^l

		/**
		 * @note EY : 20171023 remember to fix the calculation of (thread) blocks on a grid to allow for 
		 * arrays of size >> total number of threads allowed on the grid
		 * */
		Axons[idx_axon].actf(M_x); 
		Axons[idx_axon].do_Dpsi(M_x);
	}
}

/* ========== Cost functional J ========== */
float DNN::compute_costJ_L2norm() {
	const int Lp1 = sizeDimsvec.size(); // L = total number of Axons and so L+1 is total number of layers
	const int K = sizeDimsvec[Lp1-1]; // size dim. of the a^L output layer for axon L, i.e. \widehat{h}, the prediction
	const int SIZE_Y= K * m; 

	auto yhat = Axons[Lp1-2].getal(); // L+1 - 2 = L-1 which is the last axon, when counting from 0, 0,1,...L-1
	

	std::unique_ptr<hipblasHandle_t,del_cublasHandle_struct> handle_u(
		new hipblasHandle_t);
	hipblasCreate(handle_u.get());	

	// in this scope, make res to store results from taking the difference
	std::unique_ptr<float[], deleterRR_struct> res(new float[SIZE_Y], deleterRR_struct());
	hipMallocManaged((void **) &res, SIZE_Y*sizeof(float));


	/**
	 * @details C = \alpha op(A) + \beta op(B)
	 * lda - input - leading dim. of 2-dim. array used to store matrix A, lda = m 
	 * ldb - input - leading dim. of 2-dim. array used to store matrix B, ldb = m 
	 * ldc - input - leading dim. of 2-dim. array used to store matrix C, ldc = m 
	 * @note Why lda-ldb=ldc = m is because, for linear regression case, 
	 * 			yhat or \widehat{y} \in \text{Mat}_{\mathbb{R}}(m,K), it's a matrix of 
	 * m rows and K columns.  Since we're assuming COLUMN-major ordering, m is the "leading dim." 
	 * */
	float a1 = 1.0f;
	float bet = -1.0f; 
	hipblasSgeam(*handle_u.get(), 
		HIPBLAS_OP_N, HIPBLAS_OP_N, m, K, &a1, 
		yhat.get(), 
		m, &bet, y.get(), m, 
		res.get(), m );
					
	float costJ = 0.f;
	// do the L2 Euclidean norm element-wise
	hipblasSnrm2(*handle_u.get(), SIZE_Y, res.get(), 1, &costJ);
	costJ = 0.5f*costJ*costJ/((float) m) ;
	
	/** 
	 * @ref https://stackoverflow.com/questions/21589595/does-using-reset-on-a-stdshared-ptr-delete-all-instances
	 * */
	yhat.reset();
	return costJ;
	
}


float DNN::compute_costJ_xent(const int Mx) {
	const int Lp1 = sizeDimsvec.size(); // L = total number of Axons and so L+1 is total number of layers
	const int K = sizeDimsvec[Lp1-1]; // size dim. of the a^L output layer for axon L, i.e. \widehat{h}, the prediction
	const int SIZE_Y= K * m; 

	auto yhat = Axons[Lp1-2].getal(); // L+1 - 2 = L-1 which is the last axon, when counting from 0, 0,1,...L-1


	// in this scope, make res to store results from taking the so-called cross-entropy function, element-wise
	std::unique_ptr<float[], deleterRR_struct> entropys(new float[SIZE_Y], deleterRR_struct());
	hipMallocManaged((void **) &entropys, SIZE_Y*sizeof(float));

	/* ===== grid, thread block size dimensions ===== */
	// M_x = number of threads in a (single) block in x-direction
	/** 
	 * @note EY : 20171023 I will need to change this calculation of 
	 * N_x = number of (thread) blocks on a grid in x-direction
	 * so to allow for SIZE_Y >> max. allowed gridDimx.x*blockDim.x i.e. maximum allowed threads to launch on a grid
	 * */
	const int Nx = (SIZE_Y + Mx -1)/Mx;
	costJ_xent_kernel<<<Nx,Mx>>>( SIZE_Y, y.get(), yhat.get(), entropys.get() );

	// ========== now do the summation ========== 

	// create 1s array, array of 1s
	const int SIZE_ONES = SIZE_Y;
	std::unique_ptr<float[], deleterRR_struct> ones(new float[SIZE_ONES], deleterRR_struct());
	hipMallocManaged((void **) &ones, SIZE_ONES*sizeof(float));

	/* ===== grid, thread block size dimensions ===== */
		// M_x = number of threads in a (single) block in x-direction
	setconstval_kernel<<<Nx,Mx>>>(m,1.0f, ones.get() );
 
	// this is a clever way to do summation
	std::unique_ptr<hipblasHandle_t,del_cublasHandle_struct> handle_u(
		new hipblasHandle_t);
	hipblasCreate(handle_u.get());	

	float costJ = 0.f;
	hipblasSdot( *handle_u.get(), SIZE_Y, entropys.get(), 1, ones.get(), 1, &costJ);
	costJ = costJ/((float) m);

//	Axons[Lp1-2].move2al_from_ptr(yhat);
	yhat.reset();
	return costJ;
	
} 

/* ========== Gradient Descent ========== */

void DNN::grad_desc_step_logreg(  const float alpha_rate, int M_x)
{
	const int Lp1 = sizeDimsvec.size(); // L = total number of Axons and so L+1 is total number of layers
	const int K = sizeDimsvec[Lp1-1]; // size dim. of the a^L output layer for axon L, i.e. \widehat{h}, the prediction
	const int SIZE_Y=  m * K ; 

	const int s_Lm1 = sizeDimsvec[Lp1-2];

	std::unique_ptr<hipblasHandle_t,del_cublasHandle_struct> handle_u(
		new hipblasHandle_t);
	hipblasCreate(handle_u.get());	

	/* ===== grid, thread block size dimensions ===== */
	// M_x = number of threads in a (single) block in x-direction
	int Nx = (SIZE_Y + M_x - 1)/M_x; 
	if ( MAX_SIZE_1DARR < SIZE_Y ) {
		Nx = (MAX_SIZE_1DARR + M_x - 1) / M_x ; }

	// in this scope, make Delta to store results from take the partial derivative of the cross entropy function
	std::unique_ptr<float[], deleterRR_struct> Delta(new float[SIZE_Y], deleterRR_struct());
	hipMallocManaged((void **) &Delta, SIZE_Y*sizeof(float));
	
	auto yhat = Axons[Lp1-2].getal(); // L+1 - 2 = L-1 which is the last axon, when counting from 0, 0,1,...L-1
	auto ThetaL = std::move( Axons[Lp1-2].getTheta() );
	auto bL = std::move( Axons[Lp1-2].getb() );

	Deltaxent_kernel<<<Nx,M_x>>>(SIZE_Y, y.get(), yhat.get(), Delta.get()) ;

	// then do the Hadamard product with dPsi^(L)/dz^(L)
	auto dPsiLdzL = Axons[Lp1-2].getDpsil(); // L+1 - 2 = L-1 which is the last axon, when counting from 0, 0,1,...L-1
	
	HadamardMultiply_kernel<<<Nx,M_x>>>(SIZE_Y, dPsiLdzL.get(), Delta.get());
	// thus, we've calculated \Delta_i^k \odot dPsi^L/dz^L

	auto aLm1 = Axons[Lp1-2].getalm1(); // a^{L-1}
	
	float a1 = 1.0f/ ((float) m);
	float bet = 0.f;
	// \sum_{i=1}^m (a_i^{(0)})^j \Delta_i^p = \frac{ \partial J }{ \partial \Theta_j^p }

	const int SIZE_dTHETA = s_Lm1*K;
	std::unique_ptr<float[], deleterRR_struct> dTheta(new float[SIZE_dTHETA], deleterRR_struct());
	hipMallocManaged((void **) &dTheta, SIZE_dTHETA*sizeof(float));
	
	// dTheta = (1./m)*dTheta; dTheta \in \text{Mat}_{\mathbb{R}}(d,K)
	hipblasSgemm(*handle_u.get(),
		HIPBLAS_OP_T, HIPBLAS_OP_N, s_Lm1, K, m, &a1, aLm1.get(), m, Delta.get(), m , 
			&bet, dTheta.get(), s_Lm1);

	// dB = (1./m)*dB ; dB \in \mathbb{R}^K
	const int SIZE_dB = K;
	std::unique_ptr<float[], deleterRR_struct> dB(new float[SIZE_dB], deleterRR_struct());
	hipMallocManaged((void **) &dB, SIZE_dB*sizeof(float));

	// create 1s array, array of 1s
	const int SIZE_ONES = m;
	std::unique_ptr<float[], deleterRR_struct> ones(new float[SIZE_ONES], deleterRR_struct());
	hipMallocManaged((void **) &ones, SIZE_ONES*sizeof(float));

	/* ===== grid, thread block size dimensions ===== */
	// M_x = number of threads in a (single) block in x-direction
	Nx = (SIZE_ONES + M_x - 1)/M_x; 
	if ( MAX_SIZE_1DARR < SIZE_ONES ) {
		Nx = (MAX_SIZE_1DARR + M_x - 1) / M_x ; }

	setconstval_kernel<<<Nx,M_x>>>(m,1.0f, ones.get() );
 
	// this is a clever way to do summation
	hipblasSgemm(*handle_u.get(), HIPBLAS_OP_N, HIPBLAS_OP_N, 1,K,m, 
		&a1, ones.get(), 1, Delta.get(), m, 
		&bet, dB.get(), 1); 


	bet = 1.0f; 
	a1 = -1.0f * alpha_rate; 

	// actual gradient descent iteration step
	hipblasSaxpy( *handle_u.get(), SIZE_dTHETA, &a1, dTheta.get(), 1, ThetaL.get(), 1);
	hipblasSaxpy( *handle_u.get(), SIZE_dB, &a1, dB.get(), 1, bL.get(), 1);

	// return ownership of yhat,a0 back to the Feed-forward "network"
	yhat.reset(); 
	aLm1.reset(); 
	Axons[Lp1-2].move2Dpsil_from_ptr(dPsiLdzL);

	// Before returning ownership of Theta^L, b^L, use it to compute the other components of the gradient
	std::unique_ptr<float[], deleterRR_struct> DeltaLm1(new float[m*s_Lm1], deleterRR_struct());
	hipMallocManaged((void **) &DeltaLm1, m*s_Lm1*sizeof(float));
	bet = 0.0f; 
	a1 = 1.0f ; 
	/**
	 * @ref http://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-gemm
	 * @details A device array of dim lda x k with lda >= max(1,m) and lda x m otherwise
	 * 			B device array of dim ldb x n with ldb >= max(1,k) and ldb x k otherwise 
	 * 			C device array of dim ldc x n 
	 * op(A) m x k, op(B) k x n and C m x n */  
	hipblasSgemm(*handle_u.get(),
		HIPBLAS_OP_N, HIPBLAS_OP_T, m, s_Lm1, K, &a1, Delta.get(), m, ThetaL.get(), s_Lm1 , 
			&bet, DeltaLm1.get(), m );

	Axons[Lp1-2].move2Theta_from_ptr(ThetaL);
	Axons[Lp1-2].move2b_from_ptr(bL);
	
	/* =============== l=2 =============== */

	const int s_Lm2 = sizeDimsvec[Lp1-3];
	/* ===== grid, thread block size dimensions ===== */
	// M_x = number of threads in a (single) block in x-direction
	Nx = (m*s_Lm1 + M_x - 1)/M_x; 
	if ( MAX_SIZE_1DARR < m*s_Lm1 ) {
		Nx = (MAX_SIZE_1DARR + M_x - 1) / M_x ; }

	auto ThetaLm1 = std::move( Axons[Lp1-3].getTheta() );
	auto bLm1 = std::move( Axons[Lp1-3].getb() );

	// then do the Hadamard product with dPsi^(L)/dz^(L)
	auto dPsiLm1dzLm1 = Axons[Lp1-3].getDpsil(); // L+1 - 3 = L-2 which is the L-1th axon, when counting from 0, 0,1,...L-1
	
	HadamardMultiply_kernel<<<Nx,M_x>>>(m*s_Lm1, dPsiLm1dzLm1.get(), DeltaLm1.get());
	// thus, we've calculated \Delta_i^{j_{L-1}} \odot dPsi^{L-1}/dz^{L-1}

	auto aLm2 = Axons[Lp1-3].getalm1(); // a^{L-2}
	
	a1 = 1.0f/ ((float) m);
	bet = 0.f;
	
	const int SIZE_dTHETALm1 = s_Lm2*s_Lm1;
	std::unique_ptr<float[], deleterRR_struct> dThetaLm1(new float[SIZE_dTHETALm1], deleterRR_struct());
	hipMallocManaged((void **) &dThetaLm1, SIZE_dTHETALm1*sizeof(float));
	
	// dTheta = (1./m)*dTheta; dTheta \in \text{Mat}_{\mathbb{R}}(d,K)
	hipblasSgemm(*handle_u.get(),
		HIPBLAS_OP_T, HIPBLAS_OP_N, s_Lm2, s_Lm1, m, &a1, aLm2.get(), m, DeltaLm1.get(), m , 
			&bet, dThetaLm1.get(), s_Lm2);

	// dB = (1./m)*dB ; dB \in \mathbb{R}^K
	const int SIZE_dBLm1 = s_Lm1;
	std::unique_ptr<float[], deleterRR_struct> dBLm1(new float[SIZE_dBLm1], deleterRR_struct());
	hipMallocManaged((void **) &dBLm1, SIZE_dBLm1*sizeof(float));

	// this is a clever way to do summation
	hipblasSgemm(*handle_u.get(), HIPBLAS_OP_N, HIPBLAS_OP_N, 1,s_Lm1,m, 
		&a1, ones.get(), 1, DeltaLm1.get(), m, 
		&bet, dBLm1.get(), 1); 

	bet = 1.0f; 
	a1 = -1.0f * alpha_rate; 

	// actual gradient descent iteration step
	hipblasSaxpy( *handle_u.get(), SIZE_dTHETALm1, &a1, dThetaLm1.get(), 1, ThetaLm1.get(), 1);
	hipblasSaxpy( *handle_u.get(), SIZE_dBLm1, &a1, dBLm1.get(), 1, bLm1.get(), 1);

	// return ownership of yhat,a0 back to the Feed-forward "network"
	aLm2.reset(); 
	Axons[Lp1-3].move2Dpsil_from_ptr(dPsiLm1dzLm1);

	// Before returning ownership of Theta^{L-1}, b^{L-1}, use it to compute the other components of the gradient
	std::unique_ptr<float[], deleterRR_struct> DeltaLm2(new float[m*s_Lm2], deleterRR_struct());
	hipMallocManaged((void **) &DeltaLm2, m*s_Lm2*sizeof(float));
	bet = 0.0f; 
	a1 = 1.0f ; 
	/**
	 * @ref http://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-gemm
	 * @details A device array of dim lda x k with lda >= max(1,m) and lda x m otherwise
	 * 			B device array of dim ldb x n with ldb >= max(1,k) and ldb x k otherwise 
	 * 			C device array of dim ldc x n 
	 * op(A) m x k, op(B) k x n and C m x n */  
	hipblasSgemm(*handle_u.get(),
		HIPBLAS_OP_N, HIPBLAS_OP_T, m, s_Lm2, s_Lm1, &a1, DeltaLm1.get(), m, ThetaLm1.get(), s_Lm2 , 
			&bet, DeltaLm2.get(), m );

	Axons[Lp1-3].move2Theta_from_ptr(ThetaLm1);
	Axons[Lp1-3].move2b_from_ptr(bLm1);

	/* =============== END of l=2 =============== */

	/* =============== l=3,...L =============== */
	// We've updated Theta^L, b^L, Theta^{L-1}, b^{L-1}.  Time to do the others 

	/**
	 * 	@note EY : 20171031 I realized the problem of having a std::unique_ptr that points to an array 
	 * 		but we want it to point to other arrays, of a DIFFERENT size, than the previous array.  
	 * 		I will try to use std::move
	 * */
	std::unique_ptr<float[], deleterRR_struct> ptr_DeltaLmlm2; 
	ptr_DeltaLmlm2 = std::move( DeltaLm2 ) ;

	for (int l=3; l < Lp1; l++) {
		int idx_axon = (Lp1-1) -(l-1) -1; // Lp1-1=L, total number of axons L, -1 for 0-based counting; 
		const int s_Lmlm1 = sizeDimsvec[idx_axon+1]; // s_{ L -(l-1) }
		const int s_Lml = sizeDimsvec[idx_axon];	// s_{L -l }
		/* ===== grid, thread block size dimensions ===== */
		// M_x = number of threads in a (single) block in x-direction
		Nx = (m*s_Lmlm1 + M_x - 1)/M_x; 
		if ( MAX_SIZE_1DARR < m*s_Lm1 ) {
			Nx = (MAX_SIZE_1DARR + M_x - 1) / M_x ; }

		auto ThetaLmlm1 = std::move( Axons[idx_axon].getTheta() );
		auto bLmlm1 	= std::move( Axons[idx_axon].getb() );
		
		// then do the Hadamard product with dPsi^(L-(l-1))/dz^(L-(l-1))
		auto dPsiLmlm1dzLmlm1 = Axons[idx_axon].getDpsil(); 
		
		HadamardMultiply_kernel<<<Nx,M_x>>>(m*s_Lmlm1, dPsiLmlm1dzLmlm1.get(), ptr_DeltaLmlm2.get());
		// thus, we've calculated \Delta_i^{j_{L-1}} \odot dPsi^{L-1}/dz^{L-1}

		auto aLml = Axons[idx_axon].getalm1(); // a^{L-l}
	
		a1 = 1.0f/ ((float) m);
		bet = 0.f;
	
		const int SIZE_dTHETALmlm1 = s_Lml*s_Lmlm1;
		std::unique_ptr<float[], deleterRR_struct> dThetaLmlm1(new float[SIZE_dTHETALmlm1], deleterRR_struct());
		hipMallocManaged((void **) &dThetaLmlm1, SIZE_dTHETALmlm1*sizeof(float));
	
		// dTheta = (1./m)*dTheta; dTheta \in \text{Mat}_{\mathbb{R}}(d,K)
		hipblasSgemm(*handle_u.get(),
			HIPBLAS_OP_T, HIPBLAS_OP_N, s_Lml, s_Lmlm1, m, &a1, aLml.get(), m, ptr_DeltaLmlm2.get(), m , 
				&bet, dThetaLmlm1.get(), s_Lml);

		// dB = (1./m)*dB ; dB \in \mathbb{R}^K
		const int SIZE_dBLmlm1 = s_Lmlm1;
		std::unique_ptr<float[], deleterRR_struct> dBLmlm1(new float[SIZE_dBLmlm1], deleterRR_struct());
		hipMallocManaged((void **) &dBLmlm1, SIZE_dBLmlm1*sizeof(float));

		// this is a clever way to do summation
		hipblasSgemm(*handle_u.get(), HIPBLAS_OP_N, HIPBLAS_OP_N, 1,s_Lmlm1,m, 
			&a1, ones.get(), 1, ptr_DeltaLmlm2.get(), m, 
			&bet, dBLmlm1.get(), 1); 

		bet = 1.0f; 
		a1 = -1.0f * alpha_rate; 
	
		// actual gradient descent iteration step
		hipblasSaxpy( *handle_u.get(), SIZE_dTHETALmlm1, &a1, dThetaLmlm1.get(), 1, ThetaLmlm1.get(), 1);
		hipblasSaxpy( *handle_u.get(), SIZE_dBLmlm1, &a1, dBLmlm1.get(), 1, bLmlm1.get(), 1);

		// return ownership of yhat,a0 back to the Feed-forward "network"
		aLml.reset(); 
		Axons[idx_axon].move2Dpsil_from_ptr(dPsiLmlm1dzLmlm1);

		// Before returning ownership of Theta^{L-1}, b^{L-1}, use it to compute the other components of the gradient
		std::unique_ptr<float[], deleterRR_struct> DeltaLml(new float[m*s_Lml], deleterRR_struct());
		hipMallocManaged((void **) &DeltaLml, m*s_Lml*sizeof(float));
		bet = 0.0f; 
		a1 = 1.0f ; 
		/**
		 * @ref http://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-gemm
		 * @details A device array of dim lda x k with lda >= max(1,m) and lda x m otherwise
		 * 			B device array of dim ldb x n with ldb >= max(1,k) and ldb x k otherwise 
		 * 			C device array of dim ldc x n 
		 * op(A) m x k, op(B) k x n and C m x n */  
		hipblasSgemm(*handle_u.get(),
			HIPBLAS_OP_N, HIPBLAS_OP_T, m, s_Lml, s_Lmlm1, &a1, ptr_DeltaLmlm2.get(), m, ThetaLmlm1.get(), s_Lml , 
				&bet, DeltaLml.get(), m );

		Axons[idx_axon].move2Theta_from_ptr(ThetaLmlm1);
		Axons[idx_axon].move2b_from_ptr(bLmlm1);

		ptr_DeltaLmlm2 = std::move( DeltaLml ) ;

		
	}
	
	
}


/**	@fn grad_desc
 *	@param Mx - number of threads in a (single) thread block in x-direction
 * 				this is needed in the following:
 * 				in feedfwd, for addb, because we're doing "row-wise" addition of a row vector
 * 					across a matrix, 
 * 				and 
 * 				in grad_desc_step, for setconstval_kernel, to create a vector of 1's as 
 * 				a numerical trick for the usual (mathematical) Kronecker delta function	 
 * */
void DNN::grad_desc_logreg(  const int iterations, const float alpha_rate, int M_x)
{
	for (int iter=0; iter < iterations; iter++) 
	{
		feedfwd(M_x);
		grad_desc_step_logreg( alpha_rate, M_x);
		
	}
}



// destructor
DNN::~DNN() {}

/* ==================== END of Deep Neural Network class ==================== */


