#include "hip/hip_runtime.h"
/**
 * @file   : smartptr.cu
 * @brief  : Smart pointers content/source file in CUDA C++14, 
 * @author : Ernest Yeung <ernestyalumni@gmail.com>
 * @date   : 20171007  
 * @ref    :  
 * 
 * If you find this code useful, feel free to donate directly and easily at this direct PayPal link: 
 * 
 * https://www.paypal.com/cgi-bin/webscr?cmd=_donations&business=ernestsaveschristmas%2bpaypal%40gmail%2ecom&lc=US&item_name=ernestyalumni&currency_code=USD&bn=PP%2dDonationsBF%3abtn_donateCC_LG%2egif%3aNonHosted 
 * 
 * which won't go through a 3rd. party such as indiegogo, kickstarter, patreon.  
 * Otherwise, I receive emails and messages on how all my (free) material on 
 * physics, math, and engineering have helped students with their studies, 
 * and I know what it's like to not have money as a student, but love physics 
 * (or math, sciences, etc.), so I am committed to keeping all my material 
 * open-source and free, whether or not 
 * sufficiently crowdfunded, under the open-source MIT license: 
 * 	feel free to copy, edit, paste, make your own versions, share, use as you wish.  
 *  Just don't be an asshole and not give credit where credit is due.  
 * Peace out, never give up! -EY
 * 
 * */
/* 
 * COMPILATION TIP
 * nvcc -std=c++14 -lcublas -dc Feedfwd.cu -o Feedfwd.o
 * 
 * */
#include "Feedfwd.h"
/*
float computeJ_L2norm(const int m, 
	std::shared_ptr<float> & yhat, std::unique_ptr<float[], deleterRR_struct> & y) {
	
}*/

/**	@class LinReg
 * 	@brief Linear Regression  
 * */

// Constructors
LinReg::LinReg(std::vector<int> & sizeDimsvec) : 
	sizeDimsvec(sizeDimsvec) 
{
	const int Lp1 = sizeDimsvec.size(); // L=total number of Axons and so L+1 is total number of layers
	for (int l=1; l<Lp1; l++)  // l= lth Axon
	{
		int s_lm1 = sizeDimsvec[l-1];
		int s_l = sizeDimsvec[l-1];
		Axons.push_back( Axon(s_lm1,s_l) );
	}
	
}
		
// LinReg::LinReg(std::vector<Axon> & axons) : Axons(axons) {	}

// member functions
/*void LinReg::addAxon(Axon & axon) {
	Axons.push_back(axon);
}
*/

// for loading output data y 
/**
 * 	@fn load_y_from_hvec 
 * 	@brief load from host, y output data, as a std::vector<float>, column-major ordered
 * */		
void LinReg::load_y_from_hvec(std::vector<float>& h_yvec) {
	const int SIZE_Y= h_yvec.size(); 
	
	std::unique_ptr<float[], deleterRR_struct> d_y(new float[SIZE_Y], deleterRR_struct());
	y = std::move(d_y);

	hipMallocManaged((void **) &y, SIZE_Y*sizeof(float));
	
	
}

// for loading input data X into layer 0, a0, input layer
/**
 * 	@fn load_X_from_hvec
 * 	@brief load from host, X input data, as a std::vector<float>
 * 			Since we're then given m, number of examples (in dataset), 
 * 			load_X_from_hvec will then and go ahead and point (using std::shared_ptr)
 * 			output layer l-1 of Axon l-1 to input layer l-1 of Axon l
 *  @param const int m - number of examples
 * */		
void LinReg::load_X_from_hvec(std::vector<float>& h_Xvec, const int m) 
{
	const int SIZE_X = h_Xvec.size();
	const int d = sizeDimsvec[0];
	assert( SIZE_X == m*d); // check the total size dimensions is correct for "input layer" 
	
	// first Axon
	Axons[0].load_from_hXvec( h_Xvec, m);
	Axons[0].init_al(m);

	const int Lp1 = sizeDimsvec.size(); // L=total number of Axons and so L+1 is total number of layers
	
	for (int l=2;l<Lp1; l++) {
		int idx_axon = l-1; // l=2,3...L, idx_axon=1,2,...L-1

		auto tempshptr = std::move( Axons[idx_axon-1].getal() ); // temporary shared pointer, move ownership to it temporarily
		Axons[idx_axon].load_alm1_from_ptr( tempshptr );
		Axons[idx_axon].init_al(m);
		Axons[idx_axon-1].move2al_from_ptr( tempshptr); // move ownership back to al from temporary shared ptr
	}

	this->m=m; // store the number of training examples

}

void LinReg::feedfwd(int M_x) {
	const int Lp1 = sizeDimsvec.size(); // L = total number of Axons and so L+1 is total number of layers
	for (int l=1; l < Lp1;l++) {
		int idx_axon = l-1; // l=1,2,...L axons, idx_axon = 0,1,...L-1 (0-based counting for C/C++/Python
		Axons[idx_axon].rightMul();	// a^{l-1} \Theta = (a^{l-1}_i)^{j_{l-1}} \Theta_{j_{l-1}}^{j_l} =: z^l 
		Axons[idx_axon].addb(M_x);	// z^l +b = (z^l_i)^{j_l} + (b^{(l)})^{j_l} =: z^l
	}
}

float LinReg::compute_costJ_L2norm() {
	const int Lp1 = sizeDimsvec.size(); // L = total number of Axons and so L+1 is total number of layers
	const int K = sizeDimsvec[Lp1-1]; // size dim. of the a^L output layer for axon L, i.e. \widehat{h}, the prediction
	const int SIZE_Y= K * m; 

//	auto y_data = std::move( y.get() ); // y data, output data
	auto yhat = Axons[Lp1-2].getal(); // L+1 - 2 = L-1 which is the last axon, when counting from 0, 0,1,...L-1
	
	// custom deleter as a STRUCT for cublasHandle 
	struct del_cublasHandle_struct {
		void operator()(hipblasHandle_t* ptr) { hipblasDestroy(*ptr); }
	};
	
	std::unique_ptr<hipblasHandle_t,del_cublasHandle_struct> handle_u(
		new hipblasHandle_t);
	hipblasCreate(handle_u.get());	

	// in this scope, make res to store results from taking the difference
	std::unique_ptr<float[], deleterRR_struct> res(new float[SIZE_Y], deleterRR_struct());
	hipMallocManaged((void **) &res, SIZE_Y*sizeof(float));


	/**
	 * @details C = \alpha op(A) + \beta op(B)
	 * lda - input - leading dim. of 2-dim. array used to store matrix A, lda = m 
	 * ldb - input - leading dim. of 2-dim. array used to store matrix B, ldb = m 
	 * ldc - input - leading dim. of 2-dim. array used to store matrix C, ldc = m 
	 * @note Why lda-ldb=ldc = m is because, for linear regression case, 
	 * 			yhat or \widehat{y} \in \text{Mat}_{\mathbb{R}}(m,K), it's a matrix of 
	 * m rows and K columns.  Since we're assuming COLUMN-major ordering, m is the "leading dim." 
	 * */
	float a1 = 1.0f;
	float bet = -1.0f; 
	hipblasSgeam(*handle_u.get(), 
		HIPBLAS_OP_N, HIPBLAS_OP_N, m, K, &a1, 
		yhat.get(), 
//		m, &bet, y_data.get(), m, 
		m, &bet, y.get(), m, 
		res.get(), m );
					
	float costJ = 0.f;
	// do the L2 Euclidean norm element-wise
	hipblasSnrm2(*handle_u.get(), SIZE_Y, res.get(), 1, &costJ);
	costJ = 0.5f*costJ*costJ/((float) m) ;
	
	// return unique_ptr for y data ownership back
//	y = std::move( y_data);
	
	return costJ;
	
}



// destructor
LinReg::~LinReg() {}
